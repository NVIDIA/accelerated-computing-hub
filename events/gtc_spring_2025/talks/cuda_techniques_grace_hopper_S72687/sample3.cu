#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
// SPDX-FileCopyrightText: All rights reserved.
// SPDX-License-Identifier: Apache-2.0
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void kernel(int *data1, int *data2) {
  data1[threadIdx.x] = threadIdx.x;
  data2[threadIdx.x] = threadIdx.x;
}

int main() {
  int device_id;
  hipGetDevice(&device_id);

  int *data = (int *)malloc(128 * sizeof(int));
  hipMemLocation loc;
  loc.type = hipMemLocationTypeHost;
  cudaMemAdvise_v2(data, 128 * sizeof(int), hipMemAdviseSetPreferredLocation,
                   loc);
  cudaMemPrefetchAsync_v2(data, 128 * sizeof(int), loc, 0);

  int data2[128];
  hipMemLocation loc2;
  loc2.id = device_id;
  loc2.type = hipMemLocationTypeDevice;
  cudaMemAdvise_v2(data2, 128 * sizeof(int), hipMemAdviseSetPreferredLocation,
                   loc2);
  cudaMemPrefetchAsync_v2(data2, 128 * sizeof(int), loc2, 0);

  kernel<<<1, 128>>>(data, data2);
  hipDeviceSynchronize();

  for (int i = 0; i < 128; i++) {
    printf("(%d, %d)  ", data[i], data2[i]);
  }
  printf("\n");

  free(data);
  return 0;
}
