#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
// SPDX-FileCopyrightText: All rights reserved.
// SPDX-License-Identifier: Apache-2.0
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <iostream>

#include <hip/hip_runtime.h>
#include <cuda/atomic>
#include <hip/hip_runtime_api.h>

__global__ void kernel(int *data) {
  cuda::atomic_ref<int, cuda::thread_scope_system> d(*data);
  d.fetch_add(1);
}

void add(int *data, int value) {
  cuda::atomic_ref<int, cuda::thread_scope_system> d(*data);
  for (int i = 0; i < value; i++) {
    // Multiple adds here to test the coherency protocol
    d.fetch_add(1);
  }
}

int main() {
  int device_id;
  hipGetDevice(&device_id);

  int isHostNativeAtomicSupported;
  hipDeviceGetAttribute(&isHostNativeAtomicSupported,
                         hipDeviceAttributeHostNativeAtomicSupported, device_id);
  printf("hipDeviceAttributeHostNativeAtomicSupported: %d\n",
         isHostNativeAtomicSupported);

  int *data = (int *)malloc(sizeof(int));
  // hipMemLocation loc;
  // loc.type = hipMemLocationTypeHost;
  // cudaMemAdvise_v2(data, sizeof(int), hipMemAdviseSetPreferredLocation,
  // loc); cudaMemPrefetchAsync_v2(data, sizeof(int), loc, 0);
  hipMemLocation loc;
  loc.id = device_id;
  loc.type = hipMemLocationTypeDevice;
  cudaMemAdvise_v2(data, sizeof(int), hipMemAdviseSetPreferredLocation, loc);
  cudaMemPrefetchAsync_v2(data, sizeof(int), loc, 0);

  int niters = 100;
  int warmup = 2;
  int grid = 4096;
  int block = 128;

  auto start = std::chrono::high_resolution_clock::now();
  for (int iter = 0; iter < niters; iter++) {
    if (iter == warmup)
      start = std::chrono::high_resolution_clock::now();

    // Initialize to 0
    cuda::atomic_ref<int, cuda::thread_scope_system> atomic_data(*data);
    atomic_data.store(0);

    // Launch kernel atomics
    kernel<<<grid, block>>>(data);
    // Run CPU atomics
    add(data, grid);
    hipDeviceSynchronize();
  }
  auto end = std::chrono::high_resolution_clock::now();
  const std::chrono::duration<double> duration = end - start;

  std::cout << "Final value: " << *data << ", expected " << (block + 1) * grid
            << std::endl;
  std::cout << "Time: " << duration.count() * 1000. / (niters - warmup) << " milliseconds"
            << std::endl;

  free(data);
  return 0;
}
